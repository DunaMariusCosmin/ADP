#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <vector>
#include <iostream>
#include <stdio.h>
#include <float.h>
#include <ctime>

#define BLOCK_SIZE 16

/*********************** GRAPH STRUCTURE ****************************************/

// --- The graph data structure is an adjacency list.
typedef struct {

	// --- Contains the integer offset to point to the edge list for each vertex
	int *vertexArray;

	// --- Overall number of vertices
	int numVertices;

	// --- Contains the "destination" vertices each edge is attached to
	int *edgeArray;

	// --- Overall number of edges
	int numEdges;

	// --- Contains the weight of each edge
	float *weightArray;

} GraphData;



/********************** GENERATE RANDOM GRAPH **********************************/

void generateRandomGraph(GraphData *graph, int numVertices, int neighborsPerVertex) {

	graph->numVertices = numVertices;
	graph->vertexArray = (int *)malloc(graph->numVertices * sizeof(int));
	graph->numEdges = numVertices * neighborsPerVertex;
	graph->edgeArray = (int *)malloc(graph->numEdges * sizeof(int));
	graph->weightArray = (float *)malloc(graph->numEdges * sizeof(float));

	for (int i = 0; i < graph->numVertices; i++)
		graph->vertexArray[i] = i * neighborsPerVertex;

	int *tempArray = (int *)malloc(neighborsPerVertex * sizeof(int));
	for (int k = 0; k < numVertices; k++) {
		for (int l = 0; l < neighborsPerVertex; l++)
			tempArray[l] = INT_MAX;
		for (int l = 0; l < neighborsPerVertex; l++) {
			bool goOn = false;
			int temp;
			while (goOn == false) {
				goOn = true;
				temp = (rand() % graph->numVertices);
				for (int t = 0; t < neighborsPerVertex; t++)
					if (temp == tempArray[t]) goOn = false;
				if (temp == k) goOn = false;
				if (goOn == true) tempArray[l] = temp;
			}
			graph->edgeArray[k * neighborsPerVertex + l] = temp;
			graph->weightArray[k * neighborsPerVertex + l] = (float)(rand() % 99 +1) ;
		}
	}
}



/************************* ARRAY INITIALIZATION KERNER *************************/

__global__ void initializeArray( float* __restrict__ d_shortestDistances,
	const int sourceVertex, const int numVertices) {

	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < numVertices) {

		if (sourceVertex == tid) {

			d_shortestDistances[tid] = 0.f;
		}

		else {

			d_shortestDistances[tid] = FLT_MAX;
		}
	}
}




/************************ Bellman Ford CPU FUNCTION ***********************/

void bellmanFordCPU(GraphData *graph, float *h_shortestDistances, int sourceVertex, const int N) {

	// --- Initialize h_shortestDistancesances as infinite 
	for (int i = 0; i < N; i++){
			h_shortestDistances[i] = FLT_MAX;
		}


	// --- h_shortestDistancesance of the source vertex from itself is always 0
	h_shortestDistances[sourceVertex] = 0.f;

	// ---Iterations
	for (int iterCount = 0; iterCount < N - 1; iterCount++) {

	// --- Relaxation loop
		for (int j = 0; j < N; j++)
		{
			for (int l = 0; l < graph->numEdges/graph->numVertices; l++) {
				if (h_shortestDistances[ graph->edgeArray[ graph->vertexArray[j] + l ] ] > h_shortestDistances[j] + graph->weightArray[graph->edgeArray[graph->vertexArray[j] + l]]) {
					h_shortestDistances[graph->edgeArray[graph->vertexArray[j] + l]] = h_shortestDistances[j] + graph->weightArray[graph->edgeArray[graph->vertexArray[j] + l]];
				}
			}
		}
	}
}




/************************** BELLMAN FORD GPU KERNEL  **************************/

__global__  void relax(const int * __restrict__ vertexArray, const int* __restrict__ edgeArray,
	const float * __restrict__ weightArray, float* __restrict__ shortestDistances,
	 const int numVertices, const int numEdges) {

	int tid = blockIdx.x*blockDim.x + threadIdx.x;

	if (tid < numVertices) {
		for (int i = 0; i <= numEdges / numVertices; i++) {
			if (shortestDistances[edgeArray[vertexArray[tid] + i]] > shortestDistances[tid] + weightArray[edgeArray[vertexArray[tid] + i]] ) {
				shortestDistances[edgeArray[vertexArray[tid] + i]] = shortestDistances[tid] + weightArray[edgeArray[vertexArray[tid] + i]];
			}
		}
	}
}




/************************ BELLMAN FORD GPU FUNCTION *************************/

void bellmanFordGPU(GraphData *graph, const int sourceVertex, float * __restrict__ h_shortestDistances, float & elapsedGPU) {

	// --- Create device-side adjacency-list, namely, vertex array Va, edge array Ea and weight array Wa from G(V,E,W)
	int     *d_vertexArray;         
	hipMalloc(&d_vertexArray, sizeof(int)   * graph->numVertices);
	
	int     *d_edgeArray;           
	hipMalloc(&d_edgeArray, sizeof(int)   * graph->numEdges);

	float   *d_weightArray;       
	hipMalloc(&d_weightArray, sizeof(float) * graph->numEdges);

	// --- Copy adjacency-list to the device
	hipMemcpy(d_vertexArray, graph->vertexArray, sizeof(int)   * graph->numVertices, hipMemcpyHostToDevice);
	hipMemcpy(d_edgeArray, graph->edgeArray, sizeof(int)   * graph->numEdges, hipMemcpyHostToDevice);
	hipMemcpy(d_weightArray, graph->weightArray, sizeof(float) * graph->numEdges, hipMemcpyHostToDevice);

	
	float   *d_shortestDistances;          
	hipMalloc(&d_shortestDistances, sizeof(float) * graph->numVertices);
	
	// Invoke kernel 
	int threadsPerBlock = 1024; 
	int blocksPerGrid = (graph->numVertices + threadsPerBlock - 1) / threadsPerBlock;
	initializeArray <<<blocksPerGrid, threadsPerBlock >> >(d_shortestDistances,sourceVertex, graph->numVertices);
	hipPeekAtLastError();
	hipDeviceSynchronize();

	clock_t beginGPU = clock();
		for (int asyncIter = 0; asyncIter < graph->numVertices-1; asyncIter++) {
			relax << <blocksPerGrid, threadsPerBlock >> >(d_vertexArray, d_edgeArray, d_weightArray, d_shortestDistances, graph->numVertices, graph->numEdges);
			hipPeekAtLastError();
			hipDeviceSynchronize();
			
		}
	clock_t endGPU = clock();
    elapsedGPU = float(endGPU - beginGPU) / CLOCKS_PER_SEC;

	// --- Copy the result to host
	hipMemcpy(h_shortestDistances, d_shortestDistances, sizeof(float) * graph->numVertices, hipMemcpyDeviceToHost);


	hipFree(d_vertexArray);
	hipFree(d_edgeArray);
	hipFree(d_weightArray);
	hipFree(d_shortestDistances);

}


int main() {
	srand(time(NULL));
	// --- Number of graph vertices
	int numVertices = 5000;

	// --- Number of edges per graph vertex
	int neighborsPerVertex = 100;

	// --- Source vertex
	int sourceVertex = 0;

	// --- Allocate memory for arrays
	GraphData graph;
	generateRandomGraph(&graph, numVertices, neighborsPerVertex);

	// --- From adjacency list to adjacency matrix.
	//     Initializing the adjacency matrix
	//float *weightMatrix = (float *)malloc(numVertices * numVertices * sizeof(float));
	//for (int k = 0; k < numVertices * numVertices; k++) weightMatrix[k] = FLT_MAX;

	// --- Displaying the adjacency list and constructing the adjacency matrix
	/*printf("Adjacency list\n");
	for (int k = 0; k < numVertices; k++) weightMatrix[k * numVertices + k] = 0.f;
	for (int k = 0; k < numVertices; k++)
		for (int l = 0; l < neighborsPerVertex; l++) {
			weightMatrix[k * numVertices + graph.edgeArray[graph.vertexArray[k] + l]] = graph.weightArray[graph.vertexArray[k] + l];
			printf("Vertex nr. %i; Edge nr. %i; Weight = %f\n", k, graph.edgeArray[graph.vertexArray[k] + l],
				graph.weightArray[graph.vertexArray[k] + l]);
		}
		*/
	// --- Displaying the adjacency matrix
	/*printf("\nAdjacency matrix\n");
	for (int k = 0; k < numVertices; k++) {
		for (int l = 0; l < numVertices; l++)
			if (weightMatrix[k * numVertices + l] < FLT_MAX)
				printf("%1.3f\t", weightMatrix[k * numVertices + l]);
			else
				printf("--\t");
		printf("\n");
	}
	*/

	// --- Running Bellman Ford on the CPU
	float *h_shortestDistancesCPU = (float *)malloc(numVertices * sizeof(float));
	clock_t beginCPU = clock();
	bellmanFordCPU(&graph, h_shortestDistancesCPU, sourceVertex, numVertices);
	clock_t endCPU = clock();

     printf("\nCPU results\n");
	for (int k = 0; k < numVertices; k++) printf("From vertex %i to vertex %i = %f\n", sourceVertex, k, h_shortestDistancesCPU[k]);

	// --- Running Bellman Ford on the GPU
	float elapsedGPU_secs;
	float *h_shortestDistancesGPU = (float*)malloc(sizeof(float) * graph.numVertices);
	bellmanFordGPU(&graph, sourceVertex, h_shortestDistancesGPU, elapsedGPU_secs);
	printf("\nGPU results\n");
	for (int k = 0; k < numVertices; k++) printf("From vertex %i to vertex %i = %f\n", sourceVertex, k, h_shortestDistancesGPU[k]);

	// --- Calculate time
	float elapsedCPU_secs = float(endCPU - beginCPU) / CLOCKS_PER_SEC;
	printf("\nTime using CPU:       %f sec\n", elapsedCPU_secs);
	printf("Time using Cuda(GPU):   %f sec\n", elapsedGPU_secs);
	
	free(h_shortestDistancesCPU);
	free(h_shortestDistancesGPU);

	printf("Blocks per Grid used:   %d \n",(numVertices + 1024 - 1) /1024);
	return 0;
}